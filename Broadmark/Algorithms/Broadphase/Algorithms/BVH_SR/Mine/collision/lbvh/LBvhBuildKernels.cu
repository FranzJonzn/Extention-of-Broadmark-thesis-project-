#include "hip/hip_runtime.h"
#include "LBvhKernels.cuh"
#include <hip/hip_runtime.h>
#include "utility\CudaDeviceUtils.h"
#include "BvhExtNode.h"
#include "BvhIntNode.h"
#include "setting\BvhSettings.h"

namespace mn {



	__global__ void calcMC64s(int size, int3* _faces, PointType* _vertices, BOX* scene, uint64* codes) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= size) return;
		BOX bv{};
		auto v = _vertices[_faces[idx].x];
		bv.combines(v.x, v.y, v.z);
		v = _vertices[_faces[idx].y];
		bv.combines(v.x, v.y, v.z);
		v = _vertices[_faces[idx].z];
		bv.combines(v.x, v.y, v.z);
		const PointType c = bv.center();
		const PointType offset = c - scene->_min;
		codes[idx] = morton3D64(offset.x / scene->width(), offset.y / scene->height(), offset.z / scene->depth());
	}

	__global__ void copyBackCodes(int size, uint64* _primcodes, uint* _codes) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= size) return;
		_primcodes[idx] = _codes[idx] << 32;
	}


	__global__ void buildIntNodes(int size, uint *_depths, BvhExtNodeCompletePort _lvs, BvhIntNodeCompletePort _tks) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= size) return;
		_lvs.lca(idx) = -1, _depths[idx] = 0;
		int		l = idx - 1, r = idx;	///< (l, r]
		bool	mark;

		if (l >= 0)	mark = _lvs.getmetric(l) < _lvs.getmetric(r);	///< true when right child, false otherwise
		else		mark = false;

		int		cur = mark ? l : r;
		_lvs.par(idx) = cur;
		if (mark)	_tks.rc(cur) = idx, _tks.rangey(cur) = idx, atomicOr(&_tks.mark(cur), 0x00000002), _lvs.mark(idx) = 0x00000007;
		else		_tks.lc(cur) = idx, _tks.rangex(cur) = idx, atomicOr(&_tks.mark(cur), 0x00000001), _lvs.mark(idx) = 0x00000003;

		while (atomicAdd(&_tks.flag(cur), 1) == 1) {
			//_tks.update(cur, _lvs);	/// Update
			_tks.refit(cur, _lvs);	/// Refit
			_tks.mark(cur) &= 0x00000007;

			l = _tks.rangex(cur) - 1, r = _tks.rangey(cur);
			_lvs.lca(l + 1) = cur/*, _tks.rcd(cur) = ++_lvs.rcl(r)*/, _depths[l + 1]++;
			if (l >= 0)	mark = _lvs.metric(l) < _lvs.metric(r);	///< true when right child, false otherwise
			else		mark = false;

			if (l + 1 == 0 && r == size - 1) {
				_tks.par(cur) = -1;
				_tks.mark(cur) &= 0xFFFFFFFB;
				break;
			}

			int par = mark ? l : r;
			_tks.par(cur) = par;
			if (mark)	_tks.rc(par) = cur, _tks.rangey(par) = r    , atomicAnd(&_tks.mark(par), 0xFFFFFFFD), _tks.mark(cur) |= 0x00000004;
			else		_tks.lc(par) = cur, _tks.rangex(par) = l + 1, atomicAnd(&_tks.mark(par), 0xFFFFFFFE), _tks.mark(cur) &= 0xFFFFFFFB;
			cur = par;
		}
	}

	__global__ void calcIntNodeOrders(int size, BvhIntNodeCompletePort _tks, int* _lcas, uint* _depths, uint* _offsets, int* _tkMap) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= size) return;
		//for (; idx < size; idx += gridDim.x * blockDim.x) {
		int node = _lcas[idx], depth = _depths[idx], id = _offsets[idx];
		if (node != -1) {
			for (; depth--; node = _tks.lc(node)) {
				_tkMap[node] = id++;
			}
		}
		//}
	}

	__global__ void updateBvhExtNodeLinks(int size, const int *_mapTable, int* _lcas, int* _pars) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= size) return;
		int ori;
		_pars[idx] = _mapTable[_pars[idx]];
		if ((ori = _lcas[idx]) != -1)
			_lcas[idx] = _mapTable[ori] << 1;
		else
			_lcas[idx] = idx << 1 | 1;
		//if (_lvs.getrca(idx - (size - 1)) != -1)
		//	_lvs.rca(idx - (size - 1)) = _mapTable[_lvs.getrca(idx - (size - 1))] << 1;
		//else
		//	_lvs.rca(idx - (size - 1)) = idx - (size - 1) << 1 | 1;
	}

	__global__ void reorderIntNode(int intSize, const int* _tkMap, BvhIntNodeCompletePort _unorderedTks, BvhIntNodeCompletePort _tks) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= intSize) return;
		int newId = _tkMap[idx];
		uint mark = _unorderedTks.getmark(idx);

		_tks.lc(newId) = mark & 1 ? _unorderedTks.getlc(idx) : _tkMap[_unorderedTks.getlc(idx)];
		_tks.rc(newId) = mark & 2 ? _unorderedTks.getrc(idx) : _tkMap[_unorderedTks.getrc(idx)];
		_tks.mark(newId) = mark;
		mark = _unorderedTks.getpar(idx);
		_tks.par(newId) = mark != -1 ? _tkMap[mark] : -1;
		_tks.rangex(newId) = _unorderedTks.getrangex(idx);
		_tks.rangey(newId) = _unorderedTks.getrangey(idx);
		//_tks.rcd(newId) = _rcls[mark] - _unorderedTks.getrcd(idx);
		_tks.setBV(newId, _unorderedTks, idx);
	}

	__global__ void checkRestrTrunkMap(int numRtIntNode, const int* _restrQueue, const int* _tkMap, const int * _restrIntMark, int *_rtIntCount) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= numRtIntNode) return;
		int mappedVal = _tkMap[_restrQueue[idx]];
		if (atomicAdd(_rtIntCount + mappedVal, 1) != 0)
			printf("BVH_SR: \t \n\t~~%d-th index(%d) mapped int index(%d) repeated\n", idx, _restrQueue[idx], mappedVal);
		if (_restrIntMark[mappedVal] == 0)
			printf("BVH_SR: \t \n\t~~%d-th index(%d) mapped int index(%d) not requiring restructuring\n", idx, _restrQueue[idx], mappedVal);
	}
	__global__ void checkPrimmap(int size, int* _primmap, int* _cnt) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= size) return;
		int val;
		if ((val = atomicAdd(&_cnt[_primmap[idx]], 1)) != 0)
			printf("BVH_SR: \t %d-th map record(%d) wrong %d\n", idx, _primmap[idx], val);
	}
	__global__ void checkBVHIntegrity(int size, BvhExtNodeCompletePort _leaves, BvhIntNodeCompletePort _trunks, int *tag) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		//if (idx >= size) return;
		if (idx == 0)
			if (_leaves.getlca(size) != -1) {
				printf("BVH_SR: \t wrong tail sentinel.\n");
			}
		for (; idx < size; idx += gridDim.x * blockDim.x) {
			int par = _leaves.par(idx), dep = 1;
			bool right = _leaves.mark(idx) & 4;

			if (right) {
				if (_trunks.rc(par) != idx || (_trunks.mark(par) & 2) == 0 || _trunks.rangey(par) != idx || _leaves.lca(idx) & 1 == 0 || _leaves.lca(idx) / 2 != idx)
					printf("BVH_SR: \t leaf %d(as right child) is wrong. type:%d mark: %u par: %d\n", idx, 
						   _trunks.rc(par) != idx | ((_trunks.mark(par) & 2) == 0) << 1 | (_trunks.rangey(par) != idx) << 2, _leaves.mark(idx), par),
						atomicAdd(tag, 1);
			}
			else {
				if (_trunks.lc(par) != idx || (_trunks.mark(par) & 1) == 0 || _trunks.rangex(par) != idx || _leaves.lca(idx) & 1 == 1 || _trunks.rangex(_leaves.lca(idx) / 2) != idx)
					printf("BVH_SR: \t leaf %d(as left child) is wrong. type:%d mark: %u par: %d\n", idx, 
						   _trunks.lc(par) != idx | ((_trunks.mark(par) & 1) == 0) << 1 | (_trunks.rangex(par) != idx) << 2, _leaves.mark(idx), par),
						atomicAdd(tag, 1);
			}
			//if (idx == 171)
			//	printf("BVH_SR: \t %d-th primitive: mark:%o\n", idx, _leaves.mark(idx));
			while (_trunks.par(par) != -1 && *tag < 30) {
				right = _trunks.mark(par) & 4;
				//if (par + _trunks.rangey(par) - _trunks.rangex(par) - 1 != _leaves.par(_trunks.rangey(par)))
				//	printf("BVH_SR: \t \n\nsubtree %d[%d(%d), %d(%d)] is wrong.\n\n", par,
				//		_trunks.rangex(par), _leaves.par(_trunks.rangex(par)), 
				//		_trunks.rangey(par), _leaves.par(_trunks.rangey(par))), atomicAdd(tag, 1);

				if (right) {
					if (_trunks.rc(_trunks.par(par)) != par || (_trunks.mark(_trunks.par(par)) & 2) == 2 || _trunks.rangey(_trunks.par(par)) != _trunks.rangey(par)
						|| (_trunks.mark(_trunks.par(par)) & 1) == 0 && (_trunks.rangex(par) != _trunks.rangey(_trunks.lc(_trunks.par(par))) + 1)
						|| (_trunks.mark(_trunks.par(par)) & 1) == 1 && (_trunks.rangex(_trunks.par(par)) != _trunks.lc(_trunks.par(par))))
						printf("BVH_SR: \t trunk %d(as right child) %d[%d, %d] %d[%d, %d] -> %d[%d, %d].\n", par,
							   _trunks.lc(_trunks.par(par)), _trunks.rangex(_trunks.lc(_trunks.par(par))), _trunks.rangey(_trunks.lc(_trunks.par(par))),
							   par, _trunks.rangex(par), _trunks.rangey(par),
							   _trunks.par(par), _trunks.rangex(_trunks.par(par)), _trunks.rangey(_trunks.par(par))), atomicAdd(tag, 1);
					break;
				}
				//else {
				if (_trunks.lc(_trunks.par(par)) != par || (_trunks.mark(_trunks.par(par)) & 1) == 1 || _trunks.rangex(_trunks.par(par)) != _trunks.rangex(par)
					|| _trunks.par(par) + 1 != par)
					printf("BVH_SR: \t trunk %d(as left child) is wrong.\n", par), atomicAdd(tag, 1);
				//}

				//if (idx == 171)
				//	printf("BVH_SR: \t %d-th primitive: %d level\t %d-th node [%d, %d]\t mark:%o\n", 
				//		idx, dep, par, _trunks.rangex(par), _trunks.rangey(par), _trunks.mark(par));
				++dep;
				par = _trunks.par(par);
			}
			if (dep >= 32) {
				printf("BVH_SR: \t %d-th primitive depth: %d\n", idx, dep);
			}
		}
	}






///==================================================================================================================================================================
/// broadmarkIntegration
///==================================================================================================================================================================


	__global__ void calcMCs_BroadMarkEdition(int size, Aabb* _Aabb, BOX scene, uint* codes) {
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= size) return;
		auto v = _Aabb[idx];
		BOX bv(v);
		const PointType c = bv.center();
		const PointType offset = c - scene._min;
		codes[idx] = morton3D(offset.x / scene.width(), offset.y / scene.height(), offset.z / scene.depth());
	}

	__global__ void buildPrimitives_BroadMarkEdition(int size, BvhPrimitiveCompletePort _prims, int *_primMap, Aabb *_Aabb) {	///< update idx-th _bxs to idx-th leaf
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= size) return;
		int newIdx = _primMap[idx];
		auto v = _Aabb[idx];
		BOX bv(v);
		_prims.idx(newIdx) = idx;
		_prims.type(newIdx) = static_cast<uint>(ModelType::FixedDeformableType);
		_prims.setBV(newIdx, bv);
	}
}
