#include "hip/hip_runtime.h"
#include "narrow_phase.cuh"
#include "collision\LBVH\BvhBV.h"
#include "utility\CudaDeviceUtils.h"
#include "Dependencies\gProximity\cuda_intersect_tritri.h"
namespace mn {
		
	/*__global__ void simpleNarrowPhase(uint numpair, int2* _cps, const int3* _indices, const PointType* _vertices, int* _actualCpNum) {
		int	idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= numpair) return;


		for (int i = 0; i < 6; i++) {

		}


		const int2 cp = _cps[idx];
		const PointType P0 = _vertices[_indices[cp.x].x];
		const PointType P1 = _vertices[_indices[cp.x].y];
		const PointType P2 = _vertices[_indices[cp.x].z];
		const PointType Q0 = _vertices[_indices[cp.y].x];
		const PointType Q1 = _vertices[_indices[cp.y].y];
		const PointType Q2 = _vertices[_indices[cp.y].z];
		if (!triangleIntersection2(make_float3(P0.x, P0.y, P0.z), make_float3(P1.x, P1.y, P1.z), make_float3(P2.x, P2.y, P2.z),
			make_float3(Q0.x, Q0.y, Q0.z), make_float3(Q1.x, Q1.y, Q1.z), make_float3(Q2.x, Q2.y, Q2.z))) {
			_cps[idx] = { -1, -1 };
		}
		else
			atomicAggInc(_actualCpNum);
	}*/
	//FJ_BM:
	__global__ void simpleNarrowPhase_BME(uint numpair, int2* _cps, const Aabb* _aabb, int* _actualCpNum) {
		int	idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= numpair) return;

		const int2 cp = _cps[idx];
	
		const Vec3 PaMax = _aabb[cp.x].m_max;
		const Vec3 PaMin = _aabb[cp.x].m_min;	
		const Vec3 PbMax = _aabb[cp.y].m_max;
		const Vec3 PbMin = _aabb[cp.y].m_min;
		
		bool noOvelrapp = true;
		//�verlapnings kontroll kopierade fr�n Aabb 
		bool noOvelrapp2 = PaMax.m_data[0] >= PbMin.m_data[0] && PaMin.m_data[0] <= PbMax.m_data[0] &&
						   PaMax.m_data[1] >= PbMin.m_data[1] && PaMin.m_data[1] <= PbMax.m_data[1] &&
					   	   PaMax.m_data[2] >= PbMin.m_data[2] && PaMin.m_data[2] <= PbMax.m_data[2];

		//�verlappnings f�rs�k 1, kopierad fr�n AxisSweep
		for (int axis; axis < 3; ++axis) 
			if (PaMax.m_data[0] < PbMin.m_data[0] || 
				PbMax.m_data[0] < PaMin.m_data[0]) { noOvelrapp = false;}

		printf("%i   (%d,%d) \n aMAx(%d, %d,%d)\n bMax(%d, %d,%d)\n aMin(%d, %d,%d)\n bMin(%d, %d,%d)\n"
			, idx            , noOvelrapp     , noOvelrapp2
			, PaMax.m_data[0], PaMax.m_data[1], PaMax.m_data[2]
			, PaMin.m_data[0], PaMin.m_data[1], PaMin.m_data[2]
			, PbMax.m_data[0], PbMax.m_data[1], PbMax.m_data[2]
			, PbMin.m_data[0], PbMin.m_data[1], PbMin.m_data[2]);
		if (noOvelrapp) 
			atomicAggInc(_actualCpNum);
		else
			_cps[idx] = { -1, -1 };
	}
}
